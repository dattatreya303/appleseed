#include "hip/hip_runtime.h"

#include "vector_functions.h"

#include "foundation/math/vector.h"
using namespace foundation;

#include "kernel.cuh"

__global__ void sum_kernel_builtin(float3* a, float3* b, float3* c)
{
    // Global thread index.
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Perform computations in entire thread.
    c[idx] = make_float3(
        a[idx].x + b[idx].x,
        a[idx].y + b[idx].y,
        a[idx].z + b[idx].z);
}

__global__ void sum_kernel_v3f(Vector3f* a, Vector3f* b, Vector3f* c)
{
    // Global thread index.
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Perform computations in entire thread.
    c[idx] = Vector3f(
        a[idx].x + b[idx].x,
        a[idx].y + b[idx].y,
        a[idx].z + b[idx].z);
}
