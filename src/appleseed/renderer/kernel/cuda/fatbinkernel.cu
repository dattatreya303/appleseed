#include "hip/hip_runtime.h"

#include "kernel.cuh"

extern "C"
__global__ void sum_kernel(float* a, float* b, float* c, int N)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < N)
        c[idx] = a[idx] + b[idx];
}
